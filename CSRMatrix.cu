#include "CSRMatrix.cuh"

#include <cassert>
#include <iostream>

#define CHECK_CUDA_ERRORS(_where) \
    error = hipGetLastError(); \
    if (error != hipSuccess) \
        std::cout << _where << " CUDA " \
        "error: " << hipGetErrorString(error) << '\n'; \
    assert(error == hipSuccess);

CSRMatrix::CSRMatrix(const Matrix &matrix) {
    N = matrix.rows;
    hdr = static_cast<int *>(malloc((matrix.rows + 1) * sizeof(int)));
    hdr[0] = 0;

    for (int i = 0; i < matrix.rows; i++) {
        hdr[i + 1] = hdr[i];
        for (int j = 0; j < matrix.cols; j++) {
            if (matrix.data[i * matrix.cols + j]) {
                hdr[i + 1]++;
            }
        }
    }

    idx = static_cast<int *>(malloc(hdr[matrix.rows] * sizeof(int)));
    data = static_cast<half *>(malloc(hdr[matrix.rows] * sizeof(half)));

    for (int i = 0, j = 0; i < matrix.rows * matrix.cols; i++) {
        if (matrix.data[i]) {
            idx[j] = i % matrix.rows;
            data[j] = matrix.data[i];
            j++;
        }
    }
}

CSRMatrix::~CSRMatrix() {
    free(hdr);
    free(idx);
    free(data);
}

void CSRMatrix::copyToDevice(int **gpuHdr, int **gpuIdx, half **gpuData) const {
    hipError_t error;

    hipMalloc(reinterpret_cast<void **>(gpuData), hdr[N] * sizeof(half));
    CHECK_CUDA_ERRORS("CSRMatrix::copyToDevice malloc gpuData");
    hipMalloc(reinterpret_cast<void **>(gpuHdr), (N + 1) * sizeof(int));
    CHECK_CUDA_ERRORS("CSRMatrix::copyToDevice malloc gpuHdr");
    hipMalloc(reinterpret_cast<void **>(gpuIdx), hdr[N] * sizeof(int));
    CHECK_CUDA_ERRORS("CSRMatrix::copyToDevice malloc gpuIdx");
    hipMemcpy(*gpuData, data, hdr[N] * sizeof(half),
               hipMemcpyHostToDevice);
    CHECK_CUDA_ERRORS("CSRMatrix::copyToDevice gpuData");
    hipMemcpy(*gpuHdr, hdr, (N + 1) * sizeof(int),
               hipMemcpyHostToDevice);
    CHECK_CUDA_ERRORS("CSRMatrix::copyToDevice gpuHdr");
    hipMemcpy(*gpuIdx, idx, hdr[N] * sizeof(int),
               hipMemcpyHostToDevice);
    CHECK_CUDA_ERRORS("CSRMatrix::copyToDevice gpuIdx");
}
