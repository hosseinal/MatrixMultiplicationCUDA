#include "hip/hip_runtime.h"
// nvbench harness to run selected kernels from main.cu across patterns/sizes/sparsities
#include <nvbench/nvbench.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <vector>
#include <string>
#include <memory>
#include <cassert>

#include <cassert>
#include <iostream>
#include <chrono>
#include <sstream>
#include <iomanip>
#include <mma.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cuda/barrier>

#include "matrix_generator.h"
// Include implementation so templates are available in this TU (quick solution)
#include "matrix_generator.cpp"

#ifndef CEIL_DIV
#define CEIL_DIV(_a, _b) (((_a) / (_b)) + (((_a) % (_b)) > 0 ? 1 : 0))
#endif
// WMMA (tensor core) helpers live in namespace nvcuda::wmma; make the
// nested namespace available as 'wmma' via an alias so code using
// 'wmma::fragment' compiles correctly.
using namespace nvcuda;
namespace wmma = nvcuda::wmma;
extern const int BLOCK_SIZE = 16;

#include "Matrix.cuh"
#include "CSRMatrix.cuh"
#include "BCSRMatrix.cuh"
#include "HCSRMatrix.h"

// The matrix generator provides mg::Matrix<T> as a templated alias.
// We don't need to alias it here; the project's Matrix type is declared
// in "Matrix.cuh" which is included below.

// Forward-declare kernels from main.cu so this translation unit can
// call them as CUDA kernels. Signatures must match the definitions
// in main.cu. Do NOT use extern "C" here — CUDA kernel symbols are
// emitted by nvcc with device linkage and C++ linkage; adding
// extern "C" prevents the correct linkage and causes undefined
// references at link time.
__global__ void denseMatrixMul(const half *d_A, const half *d_B, float *d_C,
                               const unsigned int n) {
    const unsigned int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    // if (rowIdx < n && colIdx < n) {
    //     float tmp = 0.0f;
    //     for (int k = 0; k < n; k++) {
    //         // Accumulate results for a single element
    //         // There's no need here to use reduction  or atomic add, because this
    //         // thread is the only one accessing this location
    //         tmp += __half2float(d_A[rowIdx * n + k]) *
    //                 __half2float(d_B[k * n + colIdx]);
    //     }
    //     d_C[rowIdx * n + colIdx] = tmp;
    // }
}

__global__ void denseMatrixMulCo(const half *d_A, const half *d_B, float *d_C,
                                 const unsigned int n) {
    const unsigned int rowIdx = blockIdx.y *
        CEIL_DIV(n, gridDim.y) + threadIdx.x / n;
    const unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x % n;

    if (rowIdx < n && colIdx < n) {
        float tmp = 0.0f;
        for (int k = 0; k < n; k++) {
            tmp += __half2float(d_A[rowIdx * n + k]) * __half2float(
                d_B[k * n + colIdx]);
        }
        d_C[rowIdx * n + colIdx] = tmp;
    }
}
__global__ void denseMatrixMulTensor(const half *d_A, const half *d_B,
                                     float *d_C, const unsigned int n) {
    // Calculate which 16x16 tile this thread block handles
    const unsigned int warp_row = blockIdx.y * 16;
    const unsigned int warp_col = blockIdx.x * 16;

    if (warp_row >= n || warp_col >= n) return;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    // Accumulate over K dimension in 16x16 chunks
    for (int k = 0; k < n; k += 16) {
        wmma::load_matrix_sync(a_frag, d_A + warp_row * n + k, n);
        wmma::load_matrix_sync(b_frag, d_B + k * n + warp_col, n);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    wmma::store_matrix_sync(d_C + warp_row * n + warp_col, c_frag, n,
                            wmma::mem_row_major);
}
__global__ void sparseMatrixMult1Co(const int *hdr, const int *idx,
                                    const half *data, const half *B, float *C,
                                    const unsigned int n) {
    const unsigned int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (rowIdx < n && colIdx < n) {
        float tmp = 0.0f;
        for (int k = hdr[rowIdx]; k < hdr[rowIdx + 1]; k++) {
            tmp += __half2float(data[k]) * __half2float(
                B[idx[k] * n + colIdx]);
        }
        C[rowIdx * n + colIdx] = tmp;
    }
}

__global__ void sparseMatrixMult1(const int *hdr, const int *idx,
                                  const half *data, const half *B, float *C,
                                  const unsigned int n) {
    const unsigned int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (rowIdx < n && colIdx < n) {
        for (int k = hdr[rowIdx]; k < hdr[rowIdx + 1]; k++) {
            C[rowIdx * n + colIdx] += __half2float(data[k]) * __half2float(
                B[idx[k] * n + colIdx]);
        }
    }
}

__global__ void sparseMatrixMulTensor(const int *hdr, const int *idx,
                                      const half *data, const half *B,
                                      float *C, const unsigned int n) {
    const unsigned int warpRow = blockIdx.y * 16;
    const unsigned int warpCol = blockIdx.x * 16;

    if (warpRow >= n || warpCol >= n) return;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    for (int k = hdr[warpRow / 16]; k < hdr[warpRow / 16 + 1]; k++) {
        wmma::load_matrix_sync(a_frag, data + k * 16 * 16, 16);
        wmma::load_matrix_sync(b_frag, B + idx[k] * 16 * n + warpCol, n);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    wmma::store_matrix_sync(C + warpRow * n + warpCol, c_frag, n,
                            wmma::mem_row_major);
}

__global__ void sparseMatrixMulTensor1(const int *hdr, const int *idx,
                                      const half *data, const half *B,
                                      float *C, const unsigned int n) {
    const unsigned int warpRow = blockIdx.y * 16;
    const unsigned int warpCol = blockIdx.x * 16;

    if (warpRow >= n || warpCol >= n) return;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    wmma::fill_fragment(c_frag, 0.0f);

#pragma unroll
    for (int k = hdr[warpRow / 16]; k < hdr[warpRow / 16 + 1]; k++) {
        wmma::load_matrix_sync(a_frag, data + k * 16 * 16, 16);
        wmma::load_matrix_sync(b_frag, B + idx[k] * 16 * n + warpCol, n);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    wmma::store_matrix_sync(C + warpRow * n + warpCol, c_frag, n,
                            wmma::mem_row_major);
}
__global__ void addMatrices(float *C, const float *CPart, const unsigned int n) {
    const unsigned int rowIdx = blockIdx.y *
        CEIL_DIV(n, gridDim.y) + threadIdx.x / n;
    const unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x % n;

    if (rowIdx < n && colIdx < n) {
        C[rowIdx * n + colIdx] += CPart[rowIdx * n + colIdx];
    }
}
// Local constant to match main.cu's thread configuration
constexpr unsigned int N_THREADS = 32;

static const std::vector<std::string> patterns = {
	"random",
	"checkerboard",
	"diagonal",
	"blockdiagonal",
	"blockrandom"
};

// Helper: fill project Matrix from generated float matrix
static void fill_Matrix_from_generated(Matrix &dst, const std::vector<std::vector<float>> &src) {
	int rows = dst.rows;
	int cols = dst.cols;
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			dst.data[i * cols + j] = __float2half(src[i][j]);
		}
	}
}

// Common generation + device copy helper
struct GenDeviceBuffers {
	Matrix *matrixA = nullptr;
	Matrix *matrixB = nullptr;
	CSRMatrix *csrA = nullptr;
	BCSRMatrix *bcsrA = nullptr;
	// device pointers
	half *gpuA_half = nullptr;
	half *gpuB_half = nullptr;
	float *gpuC = nullptr;
	float *gpuCPart = nullptr;
	int *gpuCSRHdr = nullptr, *gpuCSRIdx = nullptr;
	half *gpuCSRData = nullptr;
	int *gpuBCSRHdr = nullptr, *gpuBCSRIdx = nullptr;
	half *gpuBCSRData = nullptr;

	~GenDeviceBuffers() {
		if (gpuC) hipFree(gpuC);
		if (gpuCPart) hipFree(gpuCPart);
		if (gpuA_half) hipFree(gpuA_half);
		if (gpuB_half) hipFree(gpuB_half);
		// Note: CSR/BCSR device frees are handled by their copyToDevice callers or not needed here
		delete csrA;
		delete bcsrA;
		delete matrixA;
		delete matrixB;
	}
};

static std::unique_ptr<GenDeviceBuffers> prepare_buffers(int N, double sparsity, const std::string &pattern) {
	auto out = std::make_unique<GenDeviceBuffers>();
	// Generate float matrices with generator
	auto genA = mg::generate_matrix<float>(N, N, sparsity, pattern, 16, 123);
	auto genB = mg::generate_matrix<float>(N, N, 0.0, "random", 16, 456);

	out->matrixA = new Matrix(N, N);
	out->matrixB = new Matrix(N, N);
	fill_Matrix_from_generated(*out->matrixA, genA);
	fill_Matrix_from_generated(*out->matrixB, genB);

	// Build sparse representations from matrixA
	out->csrA = new CSRMatrix(*out->matrixA);
	out->bcsrA = new BCSRMatrix(*out->matrixA);

	// Copy CSR/BCSR to device
	out->bcsrA->copyToDevice(&out->gpuBCSRHdr, &out->gpuBCSRIdx, &out->gpuBCSRData);
	out->csrA->copyToDevice(&out->gpuCSRHdr, &out->gpuCSRIdx, &out->gpuCSRData);

	size_t bytes_half = static_cast<size_t>(N) * N * sizeof(half);
	size_t bytes_float = static_cast<size_t>(N) * N * sizeof(float);
	hipMalloc(reinterpret_cast<void **>(&out->gpuA_half), bytes_half);
	hipMalloc(reinterpret_cast<void **>(&out->gpuB_half), bytes_half);
	hipMalloc(reinterpret_cast<void **>(&out->gpuC), bytes_float);
	hipMalloc(reinterpret_cast<void **>(&out->gpuCPart), bytes_float);
	hipMemcpy(out->gpuA_half, out->matrixA->data, bytes_half, hipMemcpyHostToDevice);
	hipMemcpy(out->gpuB_half, out->matrixB->data, bytes_half, hipMemcpyHostToDevice);
	hipMemset(out->gpuC, 0, bytes_float);
	hipMemset(out->gpuCPart, 0, bytes_float);

	return out;
}

// Benchmark: denseMatrixMul (naive)
static void bench_denseMatrixMul(nvbench::state &state) {
	const int N = static_cast<int>(state.get_int64("N"));
	const int sparsP = static_cast<int>(state.get_int64("SPARS"));
	const int patIdx = static_cast<int>(state.get_int64("PAT"));
	const double spars = sparsP / 100.0;

    std::cout << "Sparsity: " << spars << ", Pattern: " << patterns.at(patIdx % patterns.size()) << std::endl;

	const std::string pattern = patterns.at(patIdx % patterns.size());

	auto buf = prepare_buffers(N, spars, pattern);

    std::cout << "Sparsity: " << spars << ", Pattern: " << patterns.at(patIdx % patterns.size()) << std::endl;

	// Disable NVBench's blocking-kernel deadlock detector for this benchmark.
	// The kernel launcher synchronizes the stream explicitly and we
	// prefer to disable the deadlock timeout rather than marking the exec
	// as synchronous so measurements run uninterrupted.
	state.set_blocking_kernel_timeout(-1);

	// grid/block similar to main.cu naive kernel
	dim3 gridSize{static_cast<unsigned int>(N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0)), static_cast<unsigned int>(N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0)), 1};
	dim3 blockSize{N_THREADS, N_THREADS, 1};
    std::cout << "Sparsity: " << spars << ", Pattern: " << patterns.at(patIdx % patterns.size()) << std::endl;
	state.add_element_count(static_cast<size_t>(N) * N);
	state.exec([&](nvbench::launch &launch){
		denseMatrixMul<<<gridSize, blockSize, 0, launch.get_stream()>>>(buf->gpuA_half, buf->gpuB_half, buf->gpuC, static_cast<unsigned int>(N));
		// hipStreamSynchronize(launch.get_stream());
	});
    std::cout << "Sparsity: " << spars << ", Pattern: " << patterns.at(patIdx % patterns.size()) << std::endl;
} 

// Benchmark: denseMatrixMulTensor (wmma)
static void bench_denseMatrixMulTensor(nvbench::state &state) {
	const int N = static_cast<int>(state.get_int64("N"));
	const int sparsP = static_cast<int>(state.get_int64("SPARS"));
	const int patIdx = static_cast<int>(state.get_int64("PAT"));
	const double spars = sparsP / 100.0;
	const std::string pattern = patterns.at(patIdx % patterns.size());

	auto buf = prepare_buffers(N, spars, pattern);
	state.set_blocking_kernel_timeout(-1);

	dim3 gridSize{static_cast<unsigned int>(N / 16), static_cast<unsigned int>(N / 16), 1};
	dim3 blockSize{32, 1, 1};

	state.add_element_count(static_cast<size_t>(N) * N);
	state.exec([&](nvbench::launch &launch){
		denseMatrixMulTensor<<<gridSize, blockSize, 0, launch.get_stream()>>>(buf->gpuA_half, buf->gpuB_half, buf->gpuC, static_cast<unsigned int>(N));
		hipStreamSynchronize(launch.get_stream());
	});
}

// Benchmark: sparseMatrixMult1
static void bench_sparseMatrixMult1(nvbench::state &state) {
	const int N = static_cast<int>(state.get_int64("N"));
	const int sparsP = static_cast<int>(state.get_int64("SPARS"));
	const int patIdx = static_cast<int>(state.get_int64("PAT"));
	const double spars = sparsP / 100.0;
	const std::string pattern = patterns.at(patIdx % patterns.size());

	auto buf = prepare_buffers(N, spars, pattern);
	state.set_blocking_kernel_timeout(-1);

	dim3 gridSize{static_cast<unsigned int>(N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0)), static_cast<unsigned int>(N / N_THREADS + (N % N_THREADS > 0 ? 1 : 0)), 1};
	dim3 blockSize{N_THREADS, N_THREADS, 1};

	state.add_element_count(static_cast<size_t>(N) * N);
	state.exec([&](nvbench::launch &launch){
		sparseMatrixMult1<<<gridSize, blockSize, 0, launch.get_stream()>>>(buf->gpuCSRHdr, buf->gpuCSRIdx, buf->gpuCSRData, buf->gpuB_half, buf->gpuC, static_cast<unsigned int>(N));
		hipStreamSynchronize(launch.get_stream());
	});
}

// Benchmark: sparseMatrixMulTensor (BCSR tensor)
static void bench_sparseMatrixMulTensor(nvbench::state &state) {
	const int N = static_cast<int>(state.get_int64("N"));
	const int sparsP = static_cast<int>(state.get_int64("SPARS"));
	const int patIdx = static_cast<int>(state.get_int64("PAT"));
	const double spars = sparsP / 100.0;
	const std::string pattern = patterns.at(patIdx % patterns.size());

	auto buf = prepare_buffers(N, spars, pattern);
	state.set_blocking_kernel_timeout(-1);

	dim3 gridSize{static_cast<unsigned int>(N / 16), static_cast<unsigned int>(N / 16), 1};
	dim3 blockSize{32, 1, 1};

	state.add_element_count(static_cast<size_t>(N) * N);
	state.exec([&](nvbench::launch &launch){
		sparseMatrixMulTensor<<<gridSize, blockSize, 0, launch.get_stream()>>>(buf->gpuBCSRHdr, buf->gpuBCSRIdx, buf->gpuBCSRData, buf->gpuB_half, buf->gpuC, static_cast<unsigned int>(N));
		hipStreamSynchronize(launch.get_stream());
	});
}

// Benchmark: cuBLAS (GEMM) - no tensor ops
static void bench_cuBLAS(nvbench::state &state) {
	const int N = static_cast<int>(state.get_int64("N"));
	const int sparsP = static_cast<int>(state.get_int64("SPARS"));
	const int patIdx = static_cast<int>(state.get_int64("PAT"));
	const double spars = sparsP / 100.0;
	const std::string pattern = patterns.at(patIdx % patterns.size());

	auto buf = prepare_buffers(N, spars, pattern);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	constexpr float alpha = 1.0f;
	constexpr float beta = 0.0f;
	state.add_element_count(static_cast<size_t>(N) * N);
	state.exec([&](nvbench::launch &launch){
		hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, buf->gpuB_half, HIP_R_16F, N, buf->gpuA_half, HIP_R_16F, N, &beta, buf->gpuC, HIP_R_32F, N, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
	});
	hipblasDestroy(handle);
}

// Benchmark: cuBLAS with Tensor Cores
static void bench_cuBLAS_Tensor(nvbench::state &state) {
	const int N = static_cast<int>(state.get_int64("N"));
	const int sparsP = static_cast<int>(state.get_int64("SPARS"));
	const int patIdx = static_cast<int>(state.get_int64("PAT"));
	const double spars = sparsP / 100.0;
	const std::string pattern = patterns.at(patIdx % patterns.size());

	auto buf = prepare_buffers(N, spars, pattern);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
	constexpr float alpha = 1.0f;
	constexpr float beta = 0.0f;
	// warm up
	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, buf->gpuB_half, HIP_R_16F, N, buf->gpuA_half, HIP_R_16F, N, &beta, buf->gpuC, HIP_R_32F, N, HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

	state.add_element_count(static_cast<size_t>(N) * N);
	state.exec([&](nvbench::launch &launch){
		hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, buf->gpuB_half, HIP_R_16F, N, buf->gpuA_half, HIP_R_16F, N, &beta, buf->gpuC, HIP_R_32F, N, HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	});

	hipblasDestroy(handle);
}

// Register benches and axes
NVBENCH_BENCH(bench_denseMatrixMul).set_name("denseMatrixMul").add_int64_axis("N", {128, 256, 512}).add_int64_axis("SPARS", {50,60,70,80,90}).add_int64_axis("PAT", {0,1,2,3,4});
// NVBENCH_BENCH(bench_denseMatrixMulTensor).set_name("denseMatrixMulTensor").add_int64_axis("N", {128, 256, 512}).add_int64_axis("SPARS", {50,60,70,80,90}).add_int64_axis("PAT", {0,1,2,3,4});
// NVBENCH_BENCH(bench_sparseMatrixMult1).set_name("sparseMatrixMult1").add_int64_axis("N", {128, 256, 512}).add_int64_axis("SPARS", {50,60,70,80,90}).add_int64_axis("PAT", {0,1,2,3,4});
// NVBENCH_BENCH(bench_sparseMatrixMulTensor).set_name("sparseMatrixMulTensor").add_int64_axis("N", {128, 256, 512}).add_int64_axis("SPARS", {50,60,70,80,90}).add_int64_axis("PAT", {0,1,2,3,4});
// NVBENCH_BENCH(bench_cuBLAS).set_name("cuBLAS_GEMM").add_int64_axis("N", {128, 256, 512}).add_int64_axis("SPARS", {50,60,70,80,90}).add_int64_axis("PAT", {0,1,2,3,4});
// NVBENCH_BENCH(bench_cuBLAS_Tensor).set_name("cuBLAS_GEMM_TENSOR").add_int64_axis("N", {128, 256, 512}).add_int64_axis("SPARS", {50,60,70,80,90}).add_int64_axis("PAT", {0,1,2,3,4});


